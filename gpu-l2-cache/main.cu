#include "hip/hip_runtime.h"
#include "../MeasurementSeries.hpp"
#include "../dtime.hpp"
#include "../gpu-error.h"
#include "../gpu-metrics/gpu-metrics.hpp"
#include <iomanip>
#include <iostream>

using namespace std;

using dtype = double;

dtype *dA, *dB;

__global__ void initKernel(dtype *A, size_t N) {
  size_t tidx = blockDim.x * blockIdx.x + threadIdx.x;
  for (int idx = tidx; idx < N; idx += blockDim.x * gridDim.x) {
    A[idx] = 1.1;
  }
}

template <int N, int BLOCKSIZE>
__global__ void sumKernel(dtype *__restrict__ A, const dtype *__restrict__ B,
                          int blockRun) {
  dtype localSum = 0;

  for (int i = 0; i < N; i++) {
    int idx = blockDim.x * blockRun * i + (blockIdx.x % blockRun) * BLOCKSIZE +
              threadIdx.x;
    localSum += B[idx];
    // A[idx] = 1.23 * B[idx];
  }
  localSum *= (dtype)1.3;
  if (threadIdx.x > 1233 || localSum == (dtype)23.12)
    A[threadIdx.x] += localSum;
}

template <int N, int blockSize> dtype callKernel(int blockCount, int blockRun) {
  sumKernel<N, blockSize><<<blockCount, blockSize>>>(dA, dB, blockRun);
  GPU_ERROR(hipPeekAtLastError());
  return 0.0;
}

template <int N> void measure(int blockRun) {

  const int blockSize = 1024;

  hipDeviceProp_t prop;
  int deviceId;
  GPU_ERROR(hipGetDevice(&deviceId));
  GPU_ERROR(hipGetDeviceProperties(&prop, deviceId));
  std::string deviceName = prop.name;
  int smCount = prop.multiProcessorCount;
  int maxActiveBlocks = 0;
  GPU_ERROR(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &maxActiveBlocks, sumKernel<N, blockSize>, blockSize, 0));

  int blockCount = 100000;

  // GPU_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

  MeasurementSeries time;
  MeasurementSeries dram_read;
  MeasurementSeries dram_write;
  MeasurementSeries L2_read;
  MeasurementSeries L2_write;

  GPU_ERROR(hipDeviceSynchronize());
  for (int i = 0; i < 9; i++) {
    const size_t bufferCount = blockRun * blockSize * N + i * 128;
    GPU_ERROR(hipMalloc(&dA, bufferCount * sizeof(dtype)));
    initKernel<<<52, 256>>>(dA, bufferCount);
    GPU_ERROR(hipMalloc(&dB, bufferCount * sizeof(dtype)));
    initKernel<<<52, 256>>>(dB, bufferCount);
    GPU_ERROR(hipDeviceSynchronize());

    double t1 = dtime();
    callKernel<N, blockSize>(blockCount, blockRun);
    GPU_ERROR(hipDeviceSynchronize());
    double t2 = dtime();
    time.add(t2 - t1);

    // measureMetricsStart({"dram__bytes_read.sum", "dram__bytes_write.sum",
    //                      "lts__t_sectors_srcunit_tex_op_read.sum",
    //                      "lts__t_sectors_srcunit_tex_op_write.sum"});
    //
    // measureMetricsStart({"GL2C_MISS_sum"});

    // callKernel<N, blockSize>(blockCount, blockRun);
    // auto metrics = measureMetricStop();
    // dram_read.add(metrics[0] * 1024);
    //  dram_write.add(metrics[1]);
    //    L2_read.add(metrics[2] * 32);
    //    L2_write.add(metrics[3] * 32);

    GPU_ERROR(hipFree(dA));
    GPU_ERROR(hipFree(dB));
  }

  double blockDV = N * blockSize * sizeof(dtype);

  double bw = blockDV * blockCount / time.median() / 1.0e9;
  cout << fixed << setprecision(0) << setw(10) << blockDV / 1024 << " kB" //
       << fixed << setprecision(0) << setw(10) << blockDV * blockRun / 1024
       << " kB"                                                         //
       << setprecision(0) << setw(10) << time.median() * 1000.0 << "ms" //
       << setprecision(1) << setw(10) << time.spread() * 100 << "%"     //
       << setw(10) << bw << " GB/s   "                                  //
       << setprecision(0) << setw(6) << dram_read.median() << " GB/s "  //
       << setprecision(0) << setw(6)
       << dram_write.median() / time.median() / 1.0e9 << " GB/s " //
       << setprecision(0) << setw(6) << L2_read.median() / time.median() / 1.0e9
       << " GB/s " //
       << setprecision(0) << setw(6)
       << L2_write.median() / time.median() / 1.0e9 << " GB/s " << endl; //
}

size_t constexpr expSeries(size_t N) {
  size_t val = 20;
  for (size_t i = 0; i < N; i++) {
    val = val * 1.04 + 1;
  }
  return val;
}

int main(int argc, char **argv) {
  initMeasureMetric();
  cout << setw(13) << "data set"   //
       << setw(12) << "exec time"  //
       << setw(11) << "spread"     //
       << setw(15) << "Eff. bw\n"; //

  for (int i = 1; i < 100000; i += max(1.0, i * 0.1)) {
    measure<32>(i);
  }
}
