#include "hip/hip_runtime.h"
#include "../MeasurementSeries.hpp"
#include "../dtime.hpp"
#include "../gpu-error.h"
#include <iomanip>
#include <iostream>

using namespace std;

const int64_t max_buffer_size = 128l * 1024 * 1024 + 2;
double *dA, *dB, *dC, *dD;

#ifdef __NVCC__
const int spoilerSize = 768;
#else
const int spoilerSize = 4 * 1024;
#endif

using kernel_ptr_type = void (*)(double *A, const double *__restrict__ B,
                                 const double *__restrict__ C,
                                 const double *__restrict__ D, const size_t N,
                                 bool secretlyFalse);

template <typename T>
__global__ void init_kernel(T *A, const T *__restrict__ B,
                            const T *__restrict__ C, const T *__restrict__ D,
                            const size_t N, bool secretlyFalse) {
  __shared__ double spoiler[spoilerSize];
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tidx >= N)
    return;

  if (secretlyFalse)
    spoiler[threadIdx.x] = B[threadIdx.x];

  A[tidx] = 0.23;

  if (secretlyFalse)
    A[tidx] = spoiler[tidx];
}

template <typename T>
__global__ void read_kernel(T *A, const T *__restrict__ B,
                            const T *__restrict__ C, const T *__restrict__ D,
                            const size_t N, bool secretlyFalse) {
  __shared__ double spoiler[spoilerSize];
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tidx >= N)
    return;

  if (secretlyFalse)
    spoiler[threadIdx.x] = B[threadIdx.x];

  double temp = B[tidx];

  if (secretlyFalse || temp == 123.0)
    A[tidx] = temp + spoiler[tidx];
}

template <typename T>
__global__ void scale_kernel(T *A, const T *__restrict__ B,
                             const T *__restrict__ C, const T *__restrict__ D,
                             const size_t N, bool secretlyFalse) {
  __shared__ double spoiler[spoilerSize];
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tidx >= N)
    return;

  if (secretlyFalse)
    spoiler[threadIdx.x] = B[threadIdx.x];

  A[tidx] = B[tidx] * 1.2;

  if (secretlyFalse)
    A[tidx] = spoiler[tidx];
}

template <typename T>
__global__ void triad_kernel(T *A, const T *__restrict__ B,
                             const T *__restrict__ C, const T *__restrict__ D,
                             const size_t N, bool secretlyFalse) {
  __shared__ double spoiler[spoilerSize];
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tidx >= N)
    return;

  if (secretlyFalse)
    spoiler[threadIdx.x] = B[threadIdx.x];

  A[tidx] = B[tidx] * 1.2 + C[tidx];

  if (secretlyFalse)
    A[tidx] = spoiler[tidx];
}

template <typename T>
__global__ void stencil1d3pt_kernel(T *A, const T *__restrict__ B,
                                    const T *__restrict__ C,
                                    const T *__restrict__ D, const size_t N,
                                    bool secretlyFalse) {
  __shared__ double spoiler[spoilerSize];
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tidx >= N - 1 || tidx == 0)
    return;

  if (secretlyFalse)
    spoiler[threadIdx.x] = B[threadIdx.x];

  A[tidx] = 0.5 * B[tidx - 1] - 1.0 * B[tidx] + 0.5 * B[tidx + 1];

  if (secretlyFalse)
    A[tidx] = spoiler[tidx];
}
template <typename T>
__global__ void stencil1d5pt_kernel(T *A, const T *__restrict__ B,
                                    const T *__restrict__ C,
                                    const T *__restrict__ D, const size_t N,
                                    bool secretlyFalse) {
  __shared__ double spoiler[spoilerSize];
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tidx >= N - 2 || tidx < 2)
    return;

  if (secretlyFalse)
    spoiler[threadIdx.x] = B[threadIdx.x];

  A[tidx] = 0.25 * B[tidx - 2] + 0.25 * B[tidx - 1] - 1.0 * B[tidx] +
            0.5 * B[tidx + 1] + 0.5 * B[tidx + 2];

  if (secretlyFalse)
    A[tidx] = spoiler[tidx];
}
void measureFunc(kernel_ptr_type func, int streamCount, int blockSize,
                 int blocksPerSM) {

#ifdef __NVCC__
  int maxActiveBlocks = 0;
  int currentCarveOut = 0;
  while (maxActiveBlocks < blocksPerSM) {
    GPU_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
        func), hipFuncAttributePreferredSharedMemoryCarveout, currentCarveOut));
    GPU_ERROR(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &maxActiveBlocks, func, blockSize, 0));
    // std::cout << maxActiveBlocks << " " << currentCarveOut << "\n";
    currentCarveOut++;
  }

#else

  int maxActiveBlocks = 0;
  GPU_ERROR(hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks,
                                                          func, blockSize, 0));
  if (maxActiveBlocks > blocksPerSM)
    cout << "! " << maxActiveBlocks << " blocks per SM ";
#endif

  MeasurementSeries time;
  MeasurementSeries power;

  func<<<max_buffer_size / blockSize + 1, blockSize>>>(dA, dB, dC, dD,
                                                       max_buffer_size, false);

  for (int iter = 0; iter < 9; iter++) {
    GPU_ERROR(hipDeviceSynchronize());
    double t1 = dtime();
    GPU_ERROR(hipDeviceSynchronize());
    func<<<max_buffer_size / blockSize + 1, blockSize>>>(
        dA, dB, dC, dD, max_buffer_size, false);
    func<<<max_buffer_size / blockSize + 1, blockSize>>>(
        dA, dB, dC, dD, max_buffer_size, false);
    GPU_ERROR(hipDeviceSynchronize());
    double t2 = dtime();
    time.add((t2 - t1) / 2);
  }

  cout << fixed << setprecision(0)
       << setw(6)
       //<< time.value() * 1000 << " "
       //<< setw(5) << time.spread() * 100
       //<< "   " << setw(5) << power.median() / 1000
       << " " << setw(5)
       << streamCount * max_buffer_size * sizeof(double) / time.median() * 1e-9;
  cout.flush();
}

void measureKernels(vector<pair<kernel_ptr_type, int>> kernels, int blockSize,
                    int blocksPerSM) {
  hipDeviceProp_t prop;
  int deviceId;
  GPU_ERROR(hipGetDevice(&deviceId));
  GPU_ERROR(hipGetDeviceProperties(&prop, deviceId));
  std::string deviceName = prop.name;
  int threadsPerSM = prop.maxThreadsPerMultiProcessor;
  int threadsPerBlock = prop.maxThreadsPerBlock;

  if (blockSize * blocksPerSM > threadsPerSM || blockSize > threadsPerBlock)
    return;

  int smCount = prop.multiProcessorCount;
  cout << setw(9) << blockSize << "   " << setw(9) << smCount * blockSize
       << "  " << setw(5) << setprecision(1)
       << (float)(blockSize * blocksPerSM) / threadsPerSM * 100.0
       << " %  |  GB/s: ";

  for (auto kernel : kernels) {
    measureFunc(kernel.first, kernel.second, blockSize, blocksPerSM);
  }

  cout << "\n";
}

int main(int argc, char **argv) {
  GPU_ERROR(hipMalloc(&dA, max_buffer_size * sizeof(double)));
  GPU_ERROR(hipMalloc(&dB, max_buffer_size * sizeof(double)));
  GPU_ERROR(hipMalloc(&dC, max_buffer_size * sizeof(double)));
  GPU_ERROR(hipMalloc(&dD, max_buffer_size * sizeof(double)));

  init_kernel<<<max_buffer_size / 1024 + 1, 1024>>>(dA, dA, dA, dA,
                                                    max_buffer_size, false);
  init_kernel<<<max_buffer_size / 1024 + 1, 1024>>>(dB, dB, dB, dB,
                                                    max_buffer_size, false);
  init_kernel<<<max_buffer_size / 1024 + 1, 1024>>>(dC, dC, dC, dC,
                                                    max_buffer_size, false);
  init_kernel<<<max_buffer_size / 1024 + 1, 1024>>>(dD, dD, dD, dD,
                                                    max_buffer_size, false);
  GPU_ERROR(hipDeviceSynchronize());

  vector<pair<kernel_ptr_type, int>> kernels = {
      {init_kernel<double>, 1},         {read_kernel<double>, 1},
      {scale_kernel<double>, 2},        {triad_kernel<double>, 3},
      {stencil1d3pt_kernel<double>, 2}, {stencil1d5pt_kernel<double>, 2}};

  cout << "blockSize   threads       %occ  |                init"
       << "       read       scale     triad       3pt        5pt\n";

  // for (int blockSize = 32; blockSize <= 1024; blockSize += 32) {
  //   measureKernels(kernels, blockSize, 1);
  // }

  for (int blockSize = 32; blockSize <= 1024; blockSize += 32) {
    measureKernels(kernels, blockSize, 2);
  }

  GPU_ERROR(hipFree(dA));
  GPU_ERROR(hipFree(dB));
  GPU_ERROR(hipFree(dC));
  GPU_ERROR(hipFree(dD));
}
