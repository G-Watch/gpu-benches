#include "hip/hip_runtime.h"
#include "../MeasurementSeries.hpp"
#include "../dtime.hpp"
#include "../gpu-error.h"
#include "../metrics.cuh"
#include <iomanip>
#include <iostream>
#include <nvml.h>

using namespace std;

const int max_buffer_size = 512 * 1024 * 1024 + 2;
double *dA, *dB, *dC, *dD;

using kernel_ptr_type = void (*)(double *A, const double *__restrict__ B,
                                 const double *__restrict__ C,
                                 const double *__restrict__ D, const size_t N);

template <typename T>
__global__ void init_kernel(T *A, const T *__restrict__ B,
                            const T *__restrict__ C, const T *__restrict__ D,
                            const size_t N) {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  for (size_t i = tidx; i < N; i += blockDim.x * gridDim.x) {
    A[i] = 0.1;
  }
}

template <typename T, int unroll>
__global__ void sum_kernel(T *A, const T *__restrict__ B,
                           const T *__restrict__ C, const T *__restrict__ D,
                           const size_t N) {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
   double sum = 0.0;

  for (int64_t i = tidx; i < N;
       i += blockDim.x * gridDim.x * unroll) {
#pragma unroll
      for (int u = 0; u < unroll; u++) {
          sum += B[i + blockDim.x * gridDim.x * u];
      }
  }

  // for (int64_t i = tidx + N - unroll * blockDim.x * gridDim.x; i < N && i >= 0;
  //     i += blockDim.x * gridDim.x) {
  //  sum += B[i];
  // }

  if (tidx == 123123) {
    A[tidx] = sum;
  }
}

template <typename T>
__global__ void dot_kernel(T *A, const T *__restrict__ B,
                           const T *__restrict__ C, const T *__restrict__ D,
                           const size_t N) {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  double sum = 0.0;
  for (size_t i = tidx; i < N; i += blockDim.x * gridDim.x) {
    sum += B[i] * C[i];
  }

  if (tidx == 123123) {
    A[tidx] = sum;
  }
}

template <typename T>
__global__ void tdot_kernel(T *A, const T *__restrict__ B,
                            const T *__restrict__ C, const T *__restrict__ D,
                            const size_t N) {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  double sum = 0.0;
  for (size_t i = tidx; i < N; i += blockDim.x * gridDim.x) {
    sum += B[i] * C[i] * D[i];
  }

  if (tidx == 123123) {
    A[tidx] = sum;
  }
}
 template <typename T>
__global__ void scale_kernel(T *A, const T *__restrict__ B,
                             const T *__restrict__ C, const T *__restrict__ D,
                             const size_t N) {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  for (size_t i = tidx; i < N; i += blockDim.x * gridDim.x) {
    A[i] = 0.2 * B[i];
  }
}

 template <typename T>
__global__ void scale_kernel_no_GSL(T *A, const T *__restrict__ B,
                             const T *__restrict__ C, const T *__restrict__ D,
                             const size_t N) {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  if( tidx >= N ) return;
  A[tidx] = 0.2 * B[tidx];
}

template <typename T>
__global__ void triad_kernel(T *A, const T *__restrict__ B,
                             const T *__restrict__ C, const T *__restrict__ D,
                             const size_t N) {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  for (size_t i = tidx; i < N; i += blockDim.x * gridDim.x) {
    A[i] = B[i] + 0.2 * C[i];
  }
}

template <typename T>
__global__ void sch_triad_kernel(T *A, const T *__restrict__ B,
                                 const T *__restrict__ C,
                                 const T *__restrict__ D, const size_t N) {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  for (size_t i = tidx; i < N; i += blockDim.x * gridDim.x) {
    A[i] = B[i] + C[i] * D[i];
  }
}

void measureFunc(kernel_ptr_type func, int stream_count, int block_count,
                 int block_size) {

  MeasurementSeries time;
  MeasurementSeries power;

  size_t buffer_size = min(8192, max_buffer_size / (block_count * block_size * 16)) * block_count * block_size * 16;



  func<<<block_count, block_size>>>(dA, dB, dC, dD, buffer_size);

  nvmlDevice_t device;
  int deviceId;
  hipGetDevice(&deviceId);
  nvmlDeviceGetHandleByIndex(deviceId, &device);

  for (int iter = 0; iter < 9; iter++) {
    GPU_ERROR(hipDeviceSynchronize());
    double t1 = dtime();
    GPU_ERROR(hipDeviceSynchronize());
    func<<<block_count, block_size>>>(dA, dB, dC, dD, buffer_size);
    func<<<block_count, block_size>>>(dA, dB, dC, dD, buffer_size);
    GPU_ERROR(hipDeviceSynchronize());
    double t2 = dtime();
    time.add((t2 - t1) / 2);
  }

  // for (int iter = 0; iter < 100; iter++) {
  //   func<<<block_count, block_size>>>(dA, dB, dC, dD, buffer_size);
  //   func<<<block_count, block_size>>>(dA, dB, dC, dD, buffer_size);
  //   unsigned int p;
  //   nvmlDeviceGetPowerUsage(device, &p);
  //   power.add(p);
  //   GPU_ERROR(hipDeviceSynchronize());
  // }

  cout << fixed << setprecision(0)
       << setw(6)
       //<< time.value() * 1000 << " "
       //<< setw(5) << time.spread() * 100
       //<< "   " << setw(5) << power.median() / 1000
       << " " << setw(5)
       << stream_count * buffer_size * sizeof(double) / time.median() * 1e-9;
  cout.flush();
}

void measureKernels(vector<pair<kernel_ptr_type, int>> kernels, int block_count,
                    int block_size, int max_blocks) {
  cout << setw(9) << block_count << "   " << setw(10)
       << block_size * block_count << "  " << setw(7) << setprecision(1)
       << (double)block_count / max_blocks * 100.0 << "  |  GB/s: ";

  for (auto kernel : kernels) {
    measureFunc(kernel.first, kernel.second, block_count, block_size);
  }

  cout << "\n";
}

int main(int argc, char **argv) {
  nvmlInit();
  GPU_ERROR(hipMalloc(&dA, max_buffer_size * sizeof(double)));
  GPU_ERROR(hipMalloc(&dB, max_buffer_size * sizeof(double)));
  GPU_ERROR(hipMalloc(&dC, max_buffer_size * sizeof(double)));
  GPU_ERROR(hipMalloc(&dD, max_buffer_size * sizeof(double)));


  init_kernel<<<256, 400>>>(dB, dB, dB, dB, max_buffer_size);
    init_kernel<<<256, 400>>>(dA, dA, dA, dA, max_buffer_size);
  init_kernel<<<256, 400>>>(dC, dC, dC, dC, max_buffer_size);
  init_kernel<<<256, 400>>>(dD, dD, dD, dD, max_buffer_size);
  GPU_ERROR(hipDeviceSynchronize());

  vector<pair<kernel_ptr_type, int>> kernels = {
      {init_kernel<double>, 1},      {sum_kernel<double, 1>, 1},
       {sum_kernel<double, 2>, 1},   {sum_kernel<double, 4>, 1},
       {sum_kernel<double, 8>, 1},   {sum_kernel<double, 16>, 1},
      {dot_kernel<double>, 2},      {tdot_kernel<double>, 3},
      {scale_kernel<double>, 2},
      {triad_kernel<double>, 3}, {sch_triad_kernel<double>, 4}};

  const int block_size = 256;
  hipDeviceProp_t prop;
  int deviceId;
  GPU_ERROR(hipGetDevice(&deviceId));
  GPU_ERROR(hipGetDeviceProperties(&prop, deviceId));
  std::string deviceName = prop.name;
  int smCount = prop.multiProcessorCount;
  int maxActiveBlocks = 0;
  GPU_ERROR(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &maxActiveBlocks, kernels[0].first, block_size, 0));

  int max_blocks = maxActiveBlocks * smCount;

  cout << "    blocks     threads     %occ  |               init       sum1    "
          "   sum2  "
          "     sum4       sum8      sum16        dot       tdot      scale "
          "     triad  sch_triad\n";

   for (int block_size = 32; block_size <= 512; block_size *= 2) {
    measureKernels(kernels, 1, block_size, max_blocks);
  }

  for (int i = 2; i < smCount; i*=2) {
    measureKernels(kernels, i, block_size, max_blocks);
  }

  for (int i = smCount; i <= smCount * maxActiveBlocks; i += smCount) {
    measureKernels(kernels, i, block_size, max_blocks);
  }


   GPU_ERROR(hipFree(dA));
   GPU_ERROR(hipFree(dB));
  GPU_ERROR(hipFree(dC));
   GPU_ERROR(hipFree(dD));
}
