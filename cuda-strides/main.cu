#include "hip/hip_runtime.h"
#include "../MeasurementSeries.hpp"
#include "../dtime.hpp"
#include "../gpu-error.h"
#include <array>
#include <iomanip>
#include <iostream>
#include <unistd.h>
#ifdef __NVCC__
#include <nvml.h>
#endif
#ifdef __HIP__
#include <rocm_smi/rocm_smi.h>
#endif

using namespace std;

using dtype = double;
const int max_buffer_size = 32 * 1024 * 1024;
dtype *dA, *dB, *dC, *dD;
using kernel_ptr_type = void (*)(dtype *A, dtype *B, int zero, int one);
unsigned int gpu_clock = 0;

template <typename T>
__global__ void init_kernel(T *A, const T *__restrict__ B, const size_t N) {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  for (size_t i = tidx; i < N; i += blockDim.x * gridDim.x) {
    A[i] = T(0.1);
  }
}

template <typename T, int STRIDE, int ITERS>
__global__ void rake_kernel(T *A, T *B, int zero, int one) {
  int tidx = (threadIdx.x + blockIdx.x * blockDim.x) % 64;

  T sum = T(0.0);
  const int N = 1000;

#pragma unroll 1
  for (int n = 0; n < N; n++) {
    int ptr = tidx * STRIDE;
    for (int i = 0; i < ITERS; i++) {
      ptr += zero;
      for (int s = 0; s < min(7, STRIDE); s++) {
        sum += A[ptr + s * one] * B[ptr + s * one];
      }
    }
    if (sum == T(123.0)) {
      B[tidx] = T(tidx);
    }
  }
}

template <typename T, int XBLOCK, int PITCH>
__global__ void block_kernel(T *A, T *B, int zero, int one) {
  int tidx = (threadIdx.x + blockIdx.x * blockDim.x) % XBLOCK;
  int tidy = (threadIdx.x + blockIdx.x * blockDim.x) / XBLOCK;

  T sum = T(0.0);
  const int N = 1000;

#pragma unroll 1
  for (int n = 0; n < N; n++) {
    for (int i = 0; i < 8; i++) {
      sum +=
          A[tidy * PITCH + tidx + i * zero] * B[tidy * PITCH + tidy + i * zero];
    }

    if (sum == T(123.0)) {
      B[tidx] = T(tidx);
    }
  }
}

void measureFunc(kernel_ptr_type func, int stream_count) {

  MeasurementSeries time;

  size_t buffer_size = 8 * 1024 * 1024;
  int block_count = 1;
  for (int block_size = 64; block_size <= 1024; block_size += 64) {
    func<<<block_count, block_size>>>(dA, dB, 0, 1);

    for (int iter = 0; iter < 21; iter++) {
      GPU_ERROR(hipDeviceSynchronize());
      double t1 = dtime();
      GPU_ERROR(hipDeviceSynchronize());
      func<<<block_count, block_size>>>(dA + iter, dB + iter, 0, 1);
      func<<<block_count, block_size>>>(dA + iter, dB + iter, 0, 1);
      GPU_ERROR(hipDeviceSynchronize());
      double t2 = dtime();
      time.add(2 * stream_count * block_size * 1000 * sizeof(dtype) /
               ((t2 - t1) / 2) * 1e-9 / (gpu_clock / 1000.0));
    }
  }

  cout << fixed << setprecision(1) << " " << setw(5) << time.maxValue();
  cout.flush();
}

template <auto Start, auto End, auto Inc, class F>
constexpr void constexpr_for(F &&f) {
  if constexpr (Start < End) {
    f(std::integral_constant<decltype(Start), Start>());
    constexpr_for<Start + Inc, End, Inc>(f);
  }
}

int main(int argc, char **argv) {

  GPU_ERROR(hipMalloc(&dA, max_buffer_size * sizeof(double)));
  GPU_ERROR(hipMalloc(&dB, max_buffer_size * sizeof(double)));
  GPU_ERROR(hipMalloc(&dC, max_buffer_size * sizeof(double)));
  GPU_ERROR(hipMalloc(&dD, max_buffer_size * sizeof(double)));

  init_kernel<<<256, 400>>>(dA, dA, max_buffer_size);
  init_kernel<<<256, 400>>>(dB, dB, max_buffer_size);
  init_kernel<<<256, 400>>>(dC, dC, max_buffer_size);
  init_kernel<<<256, 400>>>(dD, dD, max_buffer_size);
  GPU_ERROR(hipDeviceSynchronize());

  int deviceId;
  GPU_ERROR(hipGetDevice(&deviceId));

  int iters = 10;
  double dt = 0;
  std::cout << "clock: ";
  while (dt < 0.3) {
    GPU_ERROR(hipDeviceSynchronize());
    double t1 = dtime();
    for (int i = 0; i < iters; i++)
      rake_kernel<dtype, 1, 8><<<1000, 1024>>>(dA, dB, 0, 1);
    usleep(10000);

#ifdef __NVCC__
    nvmlInit();
    nvmlDevice_t device;
    nvmlDeviceGetHandleByIndex(0, &device);
    nvmlDeviceGetClockInfo(device, NVML_CLOCK_SM, &gpu_clock);
#endif
#ifdef __HIP__
    rsmi_status_t ret;
    uint32_t num_devices;
    uint16_t dev_id;
    rsmi_frequencies_t clockStruct;
    ret = rsmi_init(0);
    ret = rsmi_num_monitor_devices(&num_devices);
    ret = rsmi_dev_gpu_clk_freq_get(deviceId, RSMI_CLK_TYPE_SYS, &clockStruct);
    gpu_clock = clockStruct.frequency[clockStruct.current] / 1e6;
#endif
    GPU_ERROR(hipDeviceSynchronize());
    double t2 = dtime();
    std::cout << gpu_clock << " ";
    std::cout.flush();
    dt = t2 - t1;
    iters *= 2;
  }
  std::cout << "\n";

  constexpr_for<1020, 1026, 1>([](auto i) {
    std::cout << setw(5) << i << " ";
    measureFunc(block_kernel<dtype, 1, i>, 8);
    measureFunc(block_kernel<dtype, 2, i>, 8);
    measureFunc(block_kernel<dtype, 4, i>, 8);
    measureFunc(block_kernel<dtype, 8, i>, 8);
    measureFunc(block_kernel<dtype, 16, i>, 8);
    measureFunc(block_kernel<dtype, 32, i>, 8);
    measureFunc(block_kernel<dtype, 64, i>, 8);
    std::cout << "\n";
  });
  std::cout << "\n";

  constexpr_for<1, 128, 1>([](auto i) {
    const int N = std::max(1, 8 / i);
    measureFunc(rake_kernel<dtype, i, N>, std::min(7, (int)i) * N);
    if (i % 8 == 0)
      cout << "\n";
  });

  std::cout << "\n";
  GPU_ERROR(hipFree(dA));
  GPU_ERROR(hipFree(dB));
  GPU_ERROR(hipFree(dC));
  GPU_ERROR(hipFree(dD));
}
