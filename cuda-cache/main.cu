#include "hip/hip_runtime.h"
#include "../MeasurementSeries.hpp"
#include "../dtime.hpp"
#include "../gpu-error.h"
#include "../metrics.cuh"
#include <iomanip>
#include <iostream>

using namespace std;

double *dA, *dB, *dC;

__global__ void initKernel(double *A, size_t N) {
  size_t tidx = blockDim.x * blockIdx.x + threadIdx.x;
  for (int idx = tidx; idx < N; idx += blockDim.x * gridDim.x) {
    A[idx] = 1.1;
  }
}

template <int N, int iters, int BLOCKSIZE>
__global__ void daxpyKernel(double *A, double *B, double *C) {

  double localSum = 0;
#pragma unroll(1)
  for (int iter = 0; iter < iters; iter++) {
    for (int i = 0; i < N; i += BLOCKSIZE) {
      int idx = i + threadIdx.x;
      localSum += B[idx] * C[idx];
    }
    localSum *= 1.3;
    if (threadIdx.x > 1233)
      A[threadIdx.x + blockIdx.x * blockDim.x] = 2.3;
  }
  if (threadIdx.x > 1233)
    A[threadIdx.x] += localSum;
}

template <int N, int iters, int blockSize> double callKernel(int blockCount) {
  daxpyKernel<N, iters, blockSize><<<blockCount, blockSize>>>(dA, dB, dC);
  return 0.0;
}

template <int N> void measure() {
  const int iters = 10000;

  const int blockSize = 512;

  hipDeviceProp_t prop;
  int deviceId;
  GPU_ERROR(hipGetDevice(&deviceId));
  GPU_ERROR(hipGetDeviceProperties(&prop, deviceId));
  std::string deviceName = prop.name;
  int smCount = prop.multiProcessorCount;
  int maxActiveBlocks = 0;
  GPU_ERROR(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &maxActiveBlocks, daxpyKernel<N, iters, blockSize>, blockSize, 0));

  int blockCount = smCount * 1; // maxActiveBlocks;

  MeasurementSeries time;

  GPU_ERROR(hipDeviceSynchronize());
  for (int i = 0; i < 15; i++) {
    GPU_ERROR(hipMalloc(&dA, (N + i * 128) * sizeof(double)));
    initKernel<<<52, 256>>>(dA, N + i * 128);
    GPU_ERROR(hipMalloc(&dB, (N + i * 128) * sizeof(double)));
    initKernel<<<52, 256>>>(dB, N + i * 128);
    GPU_ERROR(hipMalloc(&dC, (N + i * 128) * sizeof(double)));
    initKernel<<<52, 256>>>(dC, N + i * 128);
    GPU_ERROR(hipDeviceSynchronize());

    double t1 = dtime();
    callKernel<N, iters, blockSize>(blockCount);
    GPU_ERROR(hipDeviceSynchronize());
    double t2 = dtime();
    time.add(t2 - t1);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
  }
  GPU_ERROR(hipMalloc(&dA, N * sizeof(double)));
  initKernel<<<52, 256>>>(dA, N);
  GPU_ERROR(hipMalloc(&dB, N * sizeof(double)));
  initKernel<<<52, 256>>>(dB, N);
  GPU_ERROR(hipMalloc(&dC, N * sizeof(double)));
  initKernel<<<52, 256>>>(dC, N);

  GPU_ERROR(hipDeviceSynchronize());
  std::function<double()> measureKernelFunction =
      std::bind(callKernel<N, iters, blockSize>, blockCount);

  double dramReadBW =
      measureMetric(measureKernelFunction, "dram_read_throughput") / 1e9;
  double dramWriteBW =
      measureMetric(measureKernelFunction, "dram_write_throughput") / 1e9;

  double L2ReadBW =
      measureMetric(measureKernelFunction, "l2_read_throughput") / 1e9;

  double L2WriteBW =
      measureMetric(measureKernelFunction, "l2_write_throughput") / 1e9;

  double texReadBW =
      measureMetric(measureKernelFunction, "tex_cache_throughput") / 1e9;

  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  double blockDV = N * sizeof(double) * 2;

  double bw = blockDV * blockCount * iters / time.value() / 1.0e9;
  cout << fixed << setprecision(0) << setw(10) << blockDV / 1024 << " kB" //
       << setprecision(0) << setw(10) << time.value() * 1000.0 << "ms"    //
       << setprecision(1) << setw(10) << time.spread() * 100 << "%"       //
       << setw(10) << bw << " GB/s"                                        //
       << setw(10) << dramReadBW << " GB/s"                                //
       << setw(10) << dramWriteBW << " GB/s"                               //
       << setw(10) << L2ReadBW << " GB/s"                                  //
       << setw(10) << L2WriteBW << " GB/s"                                 //
       << setw(10) << texReadBW << " GB/s\n";
}

int main(int argc, char **argv) {
  measureMetricInit();

  cout << setw(13) << "data set" //
       << setw(12) << "exec time" //
       << setw(11) << "spread"    //
       << setw(15) << "Eff. bw"   //
       << setw(15) << "DRAM read"  //
       << setw(15) << "DRAM write" //
       << setw(15) << "L2 Read"    //
       << setw(15) << "L2 Write"   //
       << setw(15) << "Tex Read\n";

  measure<256>();
  measure<512>();
  measure<1024>();
  measure<2 * 1024>();
  measure<4 * 1024>();
  measure<8 * 1024>();
  measure<16 * 1024>();
  measure<32 * 1024>();
  measure<64 * 1024>();
}
