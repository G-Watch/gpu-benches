#include "hip/hip_runtime.h"
#include "../dtime.hpp"
#include "../gpu-error.h"
#include <algorithm>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <nvml.h>
#include <omp.h>
#include <random>
#include <sys/time.h>

using namespace std;

template <typename T>
__global__ void pchase(T *const __restrict__ buf, T *dummy_buf, int iters) {

  size_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
  T idx = 0;

  int iter = 0;
  while (iter < iters) {
    idx = buf[idx];
    if (idx == 0)
      iter++;
  }

  if (tidx > 12313) {
    dummy_buf[0] = idx;
  }
}

int main(int argc, char **argv) {

  nvmlInit();
  nvmlDevice_t device;
  nvmlDeviceGetHandleByIndex(0, &device);
  unsigned int clock = 0;

  typedef uint64_t dtype;

  const int cl_size = 2;
  const int skip_factor = 8;

  for (size_t LEN = 2; LEN < (1 << 28); LEN *= 2) {

    const int iters = max((int64_t)1, ((int64_t)1 << 16) / LEN);
    vector<dtype> order(LEN);
    dtype *buf = NULL;
    dtype *dummy_buf = NULL;

    GPU_ERROR(
        hipMallocManaged(&buf, skip_factor * cl_size * LEN * sizeof(dtype)));
    GPU_ERROR(hipMallocManaged(&dummy_buf, sizeof(dtype)));
    for (size_t i = 0; i < LEN; i++) {
      order[i] = i + 1;
    }
    order[LEN - 1] = 0;

    std::random_device rd;
    std::mt19937 g(rd());
    shuffle(begin(order), end(order) - 1, g);

    for (int cl_lane = 0; cl_lane < cl_size; cl_lane++) {
      dtype idx = 0;
      for (size_t i = 0; i < LEN; i++) {

        buf[(idx * cl_size + cl_lane) * skip_factor] =
            skip_factor *
            (order[i] * cl_size + cl_lane + (order[i] == 0 ? 1 : 0));
        idx = order[i];
      }
    }
    buf[skip_factor * (order[LEN - 2] * cl_size + cl_size - 1)] = 0;

    pchase<dtype><<<1, 32>>>(buf, dummy_buf, iters);
    nvmlDeviceGetClockInfo(device, NVML_CLOCK_SM, &clock);
    pchase<dtype><<<1, 32>>>(buf, dummy_buf, iters);
    hipDeviceSynchronize();
    double start = dtime();
    pchase<dtype><<<1, 32>>>(buf, dummy_buf, iters);
    hipDeviceSynchronize();
    double end = dtime();

    GPU_ERROR(hipGetLastError());

    double dt = end - start;
    cout << setw(5) << clock << " " //
         << setw(8) << skip_factor * LEN * cl_size * sizeof(dtype) / 1024
         << " "                                            //
         << fixed                                          //
         << setprecision(1) << setw(5) << dt * 1000 << " " //
         << setw(7) << setprecision(1)
         << (double)dt / iters / (LEN * cl_size) * clock * 1000 * 1000 << "\n";

    GPU_ERROR(hipFree(buf));
    GPU_ERROR(hipFree(dummy_buf));
  }
  cout << "\n";
}
